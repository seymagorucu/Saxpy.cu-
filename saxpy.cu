#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (id < n) // Prevents more than N operations
	{
		y[id] = a*x[id] + y[id]; 
       // printf( " y[id] %f , " , y[id] );
	}
}

void random_float(float* random, int size)
{
	for (int i=0;i<size;i++) 
	{
		random[i]=((float)rand()/(float)(RAND_MAX));
	}
}

int main(void)
{
	int N;
	float A;
	int nDevices;
	int max_threads_per_blok = 0;
	int max_grid_size = 0;
	int max_thread_blocks = 0;
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	hipGetDeviceCount(&nDevices);
	printf("cudaGetDeviceCount: %d\n", nDevices);
	printf("There are %d CUDA devices.\n", nDevices);

	for (int i = 0; i < nDevices; i++) 
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d \n", i);
		printf("Device name: %s \n ", prop.name);
		printf("Block dimensions: %d x %d  x %d \n", prop.maxThreadsDim[0],prop.maxThreadsDim[1],  prop.maxThreadsDim[2]);
		printf("Maximum number of threads per block: %d\n", prop.maxThreadsPerBlock);
		max_threads_per_blok= prop.maxThreadsPerBlock;
		printf ("Grid dimensions:  %d x %d x %d \n", prop.maxGridSize[0],  prop.maxGridSize[1],  prop.maxGridSize[2]);
		max_grid_size = prop.maxGridSize[0];

		if (max_grid_size < prop.maxGridSize[1])
		{
			max_grid_size =  prop.maxGridSize[1];
		}
		else if (max_grid_size < prop.maxGridSize[2]) 
		{
			max_grid_size = prop.maxGridSize[2];
		}
		max_thread_blocks = max_grid_size / max_threads_per_blok; // prop.maxGridSize[0] / prop.maxThreadsDim[0] for this operation used x dimension
		printf (" Maximum number of thread blocks for x  = %d \n", max_thread_blocks);
	}

	printf("Please input an N value: ");
	scanf("%d", &N);

	printf("Please input an A value: ");
	scanf("%f", &A);	

	float *h_x, *h_y, *d_x, *d_y;
	size_t size = N * sizeof(float);

	// Allocate the host input x
	h_x = (float *)malloc(size);

	// Allocate the host input y
	h_y = (float *)malloc(size);

	// Verify that allocations succeeded
	if (h_x == NULL || h_y == NULL)
	{
		fprintf(stderr, "Failed to allocate host x and y\n");
		exit(EXIT_FAILURE);
	}

    random_float(h_x, N);
    random_float(h_y, N);

	d_x = NULL;
	err = hipMalloc((void **)&d_x, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device  x (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	d_y = NULL;
	err = hipMalloc((void **)&d_y, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device  y (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy  x from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	err = hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy  y from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	int number_of_blocks = (N+1023)/1024;

	printf("Print max_threads %d \n  " , (max_thread_blocks * max_threads_per_blok) );

	if( N <= (max_thread_blocks * max_threads_per_blok)) // cannot be greater than the total number of threads
	{        
		int number_of_threads_per_block = (N/number_of_blocks);
        //This control is added to avoid missing the number of threads when integer does not give value when number is divided.
		if (N % number_of_blocks != 0 && number_of_threads_per_block < 1024)
		{
			 number_of_threads_per_block = number_of_threads_per_block+1;
		}

		if (number_of_blocks <= max_thread_blocks  )
		{
			printf (" saxpy <<<number_of_blocks = %d , number_of_threads_per_block = %d >>>\n ",number_of_blocks ,number_of_threads_per_block);
			saxpy<<<number_of_blocks ,number_of_threads_per_block >>>(N, A, d_x, d_y);

			err = hipGetLastError();

			if (err != hipSuccess)
			{
				fprintf(stderr, "Failed to launch saxpy kernel (error code %s)!\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}
		   
		}  
	}
    else
    {
        printf ("N number is too large, please enter a smaller number\n");
    }

	err = hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy  y from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
  
	err = hipFree(d_x);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device  x (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_y);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device  y (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	free(h_x);
	free(h_y);
}
